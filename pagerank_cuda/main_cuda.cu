/*
* reference: http://home.ie.cuhk.edu.hk/~wkshum/papers/pagerank.pdf 
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <tuple>
#include <vector>
#include <chrono> // timing
#include <algorithm> // sort

/* global variables */
const int N = 62586; // number of nodes
const int num_iter = 10; // number of pagerank iterations
const std::string filename = "p2p-Gnutella31.txt";
const float d = 0.85f; // damping factor. 0.85 as defined by Google
int const blocksize = 512;

typedef float trans_m_col[N];
typedef int vis_m_col[N];

void read_inputfile( vis_m_col *visited_matrix, int outgoing_table[ N ] )
{
    // Read file and build node.
    std::ifstream infile;
    infile.open( filename );

    if (infile.fail()) {
        std::cerr << "Error opeing a file" << std::endl;
        infile.close();
        exit( 1 );
    }
	
    std::string line;
    int a, b;
    while ( getline( infile, line ) )
    {
		std::istringstream iss( line );
		if ( ! ( iss >> a >> b ) ) { break; } // Format error.

    	// Process pair (a, b).
        // std::cout << a << " " << b << std::endl;
		
        visited_matrix[ a ][ b ] = 1;
        outgoing_table[ a ] += 1;
	}

	infile.close();
}

/**
 * outgoing_table, transition_matrix, visited_matrix
*/
__global__
void update_entries( trans_m_col *transition_matrix, vis_m_col *visited_matrix, int *outgoing_table, int N )
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    int const i = idx / N;
    int const j = idx % N;

    if (i < N && j < N)
    {
        if ( outgoing_table[ j ] == 0 )
        {
            // dangling node: 1 / N
            transition_matrix[ i ][ j ] = 1.0f / N;
        }
        else if ( visited_matrix[ j ][ i ] == 1 )
        {
            // if v(j, i) is visited then a(ij) = 1/L(j)
            transition_matrix[ i ][ j ] = 1.0f / outgoing_table[ j ];
        }
        // else{ table->ij_entries_matrix[ i ][ j ] = 0.0; }
    }

}

__global__
void pagerank( float *score_table, float *old_score_table, trans_m_col *transition_matrix, float d, int N )
{
    int const j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < N)
    {
        /* update pagerank scores */
        float sum = 0.0f;

        for ( auto k = 0; k < N; ++k )
        {
            sum += old_score_table[ k ] * transition_matrix[ j ][ k ];
        }
        score_table[ j ] = d * old_score_table[ j ] + ( 1.0f - d ) * sum;
    }
}

int comp( std::tuple< int, float > const &i, std::tuple< int, float > const &j )
{
    return std::get< 1 >( i ) > std::get< 1 >( j );
}

void print_top_5( float arr[ N ] )
{
    std::vector< std::tuple< int, float > > sorted = {};
    for ( auto i = 0; i < N; ++i )
    {
        sorted.push_back( std::tuple< int, float >{ i, arr[ i ] } );
    }
    std::sort( sorted.begin(), sorted.end(), comp );
    for ( auto i = 0; i < std::min( 5, N); ++i )
    {
        std::cout << std::get< 0 >( sorted[ i ] ) << "(" << std::get< 1 >( sorted[ i ] ) << ") ";
    }
    std::cout << std::endl;
}

void print_total( float arr[ N ] )
{
    float sum = 0.0f;
    for ( auto i = 0; i < N; ++i )
    {
        sum += arr[ i ];
    }
    std::cout << "s=" << sum << std::endl;
}

int main()
{
    size_t score_t_size = N * sizeof(float);
    size_t out_t_size = N * sizeof(int);
    size_t vis_m_size = N * N * sizeof(int);
    size_t trans_m_size = N * N * sizeof(float);

    vis_m_col *visited_matrix;
    visited_matrix = ( vis_m_col * )malloc( vis_m_size );
    memset(visited_matrix, 0, vis_m_size);

    trans_m_col *transition_matrix;
    transition_matrix = ( trans_m_col * )malloc( trans_m_size );
    memset(transition_matrix, 0, trans_m_size);

    float score_table[ N ] = { 0 };
    std::fill_n(score_table, N, 1.0f / N );
    int outgoing_table[ N ] = { 0 };

    read_inputfile( visited_matrix, outgoing_table );
    
    float *dev_score_table, *dev_old_score_table;
    int *dev_outgoing_table;
    vis_m_col *dev_visited_matrix;
    trans_m_col *dev_transition_matrix;

    hipMalloc( &dev_score_table, score_t_size );
    hipMalloc( &dev_old_score_table, score_t_size );
    hipMalloc( &dev_outgoing_table, out_t_size );
    hipMalloc( &dev_visited_matrix, vis_m_size );
    hipMalloc( &dev_transition_matrix, trans_m_size );
    
    hipMemcpy( dev_score_table, score_table, score_t_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_outgoing_table, outgoing_table, out_t_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_visited_matrix, visited_matrix, vis_m_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_transition_matrix, transition_matrix, trans_m_size, hipMemcpyHostToDevice );

    auto num_blocks = ceil( N * N / static_cast< float >( blocksize ) );
    /* timing the pre processing */
    auto start_time = std::chrono::steady_clock::now();
    update_entries<<< num_blocks, blocksize >>>( dev_transition_matrix, dev_visited_matrix, dev_outgoing_table, N );
    auto end_time = std::chrono::steady_clock::now();
    auto const update_duration = std::chrono::duration_cast< std::chrono::microseconds >( end_time - start_time ).count();
    
    num_blocks = ceil( N / static_cast< float >( blocksize ) );
    /* timing the pagerank algorithm */
    start_time = std::chrono::steady_clock::now();
    /* iterations must be serial */
    for ( auto i = 0; i < num_iter - 1; ++i )
    {
        /* scores from previous iteration */
        hipMemcpy( dev_old_score_table, dev_score_table, score_t_size, hipMemcpyDeviceToDevice );
        pagerank<<< num_blocks, blocksize >>>( dev_score_table, dev_old_score_table, dev_transition_matrix, d, N );
    }
    hipMemcpy(score_table, dev_score_table, score_t_size, hipMemcpyDeviceToHost);
    end_time = std::chrono::steady_clock::now();
    auto const pr_duration = std::chrono::duration_cast< std::chrono::microseconds >( end_time - start_time ).count();

    hipFree( dev_score_table );
    hipFree( dev_old_score_table );
    hipFree( dev_outgoing_table );
    hipFree( dev_visited_matrix );
    hipFree( dev_transition_matrix );

    print_top_5( score_table );
    print_total( score_table );

    std::cout << "Entries update time = "
              << update_duration
              << " us"
              << "\nCalculation time = "
		      << pr_duration
		      << " us" 
              << "\nTotal time = "
              << update_duration + pr_duration
              << " us"
              << std::endl;
    
    return 0;
}