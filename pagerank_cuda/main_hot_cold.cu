#include "hip/hip_runtime.h"
/*
 * reference: http://home.ie.cuhk.edu.hk/~wkshum/papers/pagerank.pdf 
 */
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <tuple>
#include <chrono> // timing
#include <numeric> // accumulate
#include <algorithm> // sort

#include "pagerank_hot_cold.hpp"

using namespace csc586_matrix;
using namespace csc586_matrix::soa_matrix;

/* global variables */
const std::string filename = "p2p-Gnutella04.txt";
const int N = 10876; // number of nodes
const int num_iter = 10; // number of pagerank iterations
const float d = 0.85f; // damping factor. 0.85 as defined by Google
const int blocksize = 512;

void print_scores( Matrix_soa *table )
{
    /* Print score matrix. */
    float sum = 0;
    for ( auto i = 0; i < N; ++i )
    {
        sum += table->hot[ i ].score;
        std::cout << i << "=" << table->hot[ i ].score << std::endl;
    }
    std::cout << "s=" << sum << std::endl;
}

void print_score_sum( Matrix_soa *table )
{
    float sum = 0;
    for ( auto i = 0; i < N; ++i )
    {
        sum += table->hot[ i ].score;
    }
    std::cout << "s=" << sum << std::endl;
}

int comp( std::tuple< int, Score > const &i, std::tuple< int, Score > const &j )
{
    return std::get< 1 >( i ) > std::get< 1 >( j );
}

void print_top_5( Matrix_soa *table )
{
    std::vector< std::tuple< int, Score > > sorted = {};
    for ( auto i = 0; i < N; ++i )
    {
        sorted.push_back( std::tuple< int, Score >{ i, table->hot[ i ].score } );
    }
    std::sort( sorted.begin(), sorted.end(), comp );
    for ( auto i = 0; i < std::min( 5, N); ++i )
    {
        std::cout << std::get< 0 >( sorted[ i ] ) << "(" << std::get< 1 >( sorted[ i ] ) << ") ";
    }
    std::cout << std::endl;
}

void print_table( Matrix_soa *table )
{
    /* Print visited matrix. */
    std::cout << "Entry Visited Table: " << std::endl;
    for ( auto i = 0; i < N; ++i )
    {
        for ( auto j = 0; j < N; ++j )
        {
            std::cout << table->cold[ i ].visited_col[ j ] << " ";
        }
        std::cout << std::endl;
    }
    /* Print entry matrix. */
    std::cout << "Entry Matrix Table: " << std::endl;
    for ( auto i = 0; i < N; ++i )
    {
        for ( auto j = 0; j < N; ++j )
        {
            std::cout << table->hot[ i ].entries_col[ j ] << " ";
        }
        std::cout << std::endl;
    }
    /* Print score matrix. */
    std::cout << "Score Matrix Table: " << std::endl;
    for ( auto i = 0; i < N; ++i )
    {
        std::cout << table->hot[ i ].score << std::endl;
    }
    /* Print num entries matrix. */
    std::cout << "Entries Matrix Table: " << std::endl;
    for ( auto i = 0; i < N; ++i )
    {
        std::cout << table->cold[ i ].num_entry << std::endl;
    }
}

void read_inputfile( Matrix_soa *table )
{
    /* Read file and build node. */
    std::ifstream infile;
    infile.open( filename );

    if (infile.fail()) {
        std::cerr << "Error opeing a file" << std::endl;
        infile.close();
        exit( 1 );
    }
	
    std::string line;
    uint32_t a, b;
    while ( getline( infile, line ) )
    {
		std::istringstream iss( line );
		if ( ! ( iss >> a >> b ) ) { break; } // Format error.

    	/* Process pair (a, b). */
        // std::cout << a << " " << b << std::endl;
		
        table->cold[ a ].visited_col[ b ] = 1;
        table->cold[ a ].num_entry += 1;
	}

	infile.close();
}

__global__
void update_entries( Matrix_soa *table, int N )
{
    int const i = threadIdx.x + blockIdx.x * blockDim.x;

    for ( auto j = 0; j < N; ++j )
    {
        table->hot[ i ].entries_col[ j ] = 1.0f / N;
    //     if ( table->cold[ j ].num_entry == 0 )
    //     {
    //         /* Dangling node: 1 / N */
    //         table->hot[ i ].entries_col[ j ] = 1.0f / N;
    //     }
    //     else if ( table->cold[ j ].visited_col[ i ] == 1 )
    //     {
    //         /* If v(j, i) is visited then a(ij) = 1/L(j). */
    //         table->hot[ i ].entries_col[ j ] = 1.0f / table->cold[ j ].num_entry;
    //     }
    //     // else{ table->ij_entries_matrix[ i ][ j ] = 0.0; }
    }
}

// __global__
// void cal_pagerank( Matrix_soa *table, std::vector< Score > *old_scores, int N )
// {
//     int const i = threadIdx.x + blockIdx.x * blockDim.x;
    
//     if( i < N )
//     {
//         old_scores[ i ] = table->hot[ i ].score;

//          Update pagerank scores. 
//         float sum = 0.0f;
//         for ( auto j = 0; j < N; ++j )
//         {
//             sum += old_scores[ j ] * table->hot[ i ].entries_col[ j ];
//         }
//         table->hot[ i ].score = d * old_scores[ i ] + ( 1.0f - d ) * sum;
//     }
// }

int main ()
{
    /* Initialize matrix table. */
    Matrix_soa* t = new Matrix_soa( { 
        std::vector< Tables_Hot > ( N, Tables_Hot( { 
            ( 1.0f / N ), 
            std::vector< Entry > ( N, 0.0f ) } ) ), 
        std::vector< Tables_Cold > ( N, Tables_Cold( { 
            0, 
            std::vector< Count > ( N, 0 ) } ) )
    } );

    /* Initialize Scores from previous iteration. */
    std::vector< Score > old_scores = {};
    old_scores.reserve( N );

    read_inputfile( t );

    auto const num_blocks = ceil( N / static_cast< float >( blocksize ) );

    /* Initialize pointers to memory on GPU. */
    std::vector< Score > *dev_old_scores;
    Matrix_soa *dev_table;

    /* Allocate GPU memory and copy t to there. */
    hipMalloc( (void **) &dev_table, sizeof( t ) );
    hipMalloc( (void **) &dev_old_scores, sizeof( old_scores ) );
    hipMemcpy( dev_table, t, sizeof( t ), hipMemcpyHostToDevice );
    hipMemcpy( dev_old_scores, old_scores, sizeof( old_scores ), hipMemcpyHostToDevice );
    
    update_entries<<< num_blocks, blocksize >>>( dev_table, N );
    // /* Time the pre-processing. */
    // auto start_time = std::chrono::steady_clock::now();
    // update_entries( t );
    // auto end_time = std::chrono::steady_clock::now();
    // auto const update_duration = std::chrono::duration_cast< std::chrono::microseconds >( end_time - start_time ).count();
    
    // /* Time the pagerank algorithm. */
    // start_time = std::chrono::steady_clock::now();

    // for ( auto i = 0; i < num_iter-1; ++i )
    // {
    //     cal_pagerank<<< num_blocks, blocksize >>>( dev_table, dev_old_scores, N );
    // }

    // end_time = std::chrono::steady_clock::now();
    // auto const pr_duration = std::chrono::duration_cast< std::chrono::microseconds >( end_time - start_time ).count();
    
    // /* Free GPU memory */
    hipFree( dev_table );
    hipFree( dev_old_scores);

    // // print_scores( t );
    // print_top_5( t );
    // print_score_sum( t );
    // std::cout << "Entries update time = "
    //           << update_duration
    //           << " us"
    //           << "\nCalculation time = "
		  //     << pr_duration
		  //     << " us" 
    //           << "\nTotal time = "
    //           << update_duration + pr_duration
    //           << " us"
    //           << std::endl;
    // return 0;
}