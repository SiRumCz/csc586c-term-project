/*
* reference: http://home.ie.cuhk.edu.hk/~wkshum/papers/pagerank.pdf 
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <tuple>
#include <vector>
#include <chrono> // timing
#include <algorithm> // sort

/* global variables, this is where you would change the parameters */
const long long N = 10876; // number of nodes
const int num_iter = 10; // number of pagerank iterations
const std::string filename = "../tests/p2p-Gnutella04.txt";
const float d = 0.85f; // damping factor. 0.85 as defined by Google
const int blocksize = 512;

typedef float trans_m_col[N];
typedef int vis_m_col[N];

void read_inputfile( vis_m_col *visited_matrix, int outgoing_table[ N ] )
{
    // Read file and build node.
    std::ifstream infile;
    infile.open( filename );

    if (infile.fail()) {
        std::cerr << "Error opeing a file" << std::endl;
        infile.close();
        exit( 1 );
    }
    
    std::string line;
    int a, b;
    int count_edge = 0;
    while ( getline( infile, line ) )
    {
        std::istringstream iss( line );
        if ( ! ( iss >> a >> b ) ) { break; } // Format error.

        // Process pair (a, b).
        // std::cout << a << " " << b << std::endl;
        
        visited_matrix[ a ][ b ] = 1;
        outgoing_table[ a ] += 1;

        count_edge++;
    }

    infile.close();
}

/**
 * outgoing_table, transition_matrix, visited_matrix
*/
__global__
void update_entries( trans_m_col *transition_matrix, vis_m_col *visited_matrix, int *outgoing_table, int N )
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    int const i = idx / N;
    int const j = idx % N;

    if (i < N && j < N)
    {
        if ( outgoing_table[ j ] == 0 )
        {
            // dangling node: 1 / N
            transition_matrix[ i ][ j ] = 1.0f / N;
        }
        else if ( visited_matrix[ j ][ i ] == 1 )
        {
            // if v(j, i) is visited then a(ij) = 1/L(j)
            transition_matrix[ i ][ j ] = 1.0f / outgoing_table[ j ];
        }
        // else{ table->ij_entries_matrix[ i ][ j ] = 0.0; }
    }

}

__global__
void pagerank( float *score_table, float *old_score_table, trans_m_col *transition_matrix, float d, int N )
{
    int const j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < N)
    {
        /* update pagerank scores */
        float sum = 0.0f;

        for ( auto k = 0; k < N; ++k )
        {
            sum += old_score_table[ k ] * transition_matrix[ j ][ k ];
        }
        score_table[ j ] = d * old_score_table[ j ] + ( 1.0f - d ) * sum;
    }
}

int comp( std::tuple< int, float > const &i, std::tuple< int, float > const &j )
{
    return std::get< 1 >( i ) > std::get< 1 >( j );
}

void print_top_5( float arr[ N ] )
{
    std::vector< std::tuple< int, float > > sorted = {};
    for ( auto i = 0; i < N; ++i )
    {
        sorted.push_back( std::tuple< int, float >{ i, arr[ i ] } );
    }
    std::sort( sorted.begin(), sorted.end(), comp );
    
    for ( auto i = 0; i < std::min( ( long long ) 5, N); ++i )
    {
        std::cout << std::get< 0 >( sorted[ i ] ) << "(" << std::get< 1 >( sorted[ i ] ) << ") ";
    }
    std::cout << std::endl;
}

void print_total( float arr[] )
{
    float sum = 0.0f;
    for ( auto i = 0; i < N; ++i )
    {
        sum += arr[ i ];
    }
    std::cout << "sum=" << sum << std::endl;
}

int main()
{
    auto const total_start_time = std::chrono::steady_clock::now();
    auto const score_t_size = N * sizeof(float);
    auto const out_t_size = N * sizeof(int);
    auto const vis_m_size = N * N * sizeof(int);
    auto const trans_m_size = N * N * sizeof(float);

    vis_m_col *visited_matrix;
    visited_matrix = ( vis_m_col * )malloc( vis_m_size );
    memset(visited_matrix, 0, vis_m_size);

    trans_m_col *transition_matrix;
    transition_matrix = ( trans_m_col * )malloc( trans_m_size );
    memset(transition_matrix, 0, trans_m_size);

    float score_table[ N ] = { 0 };
    std::fill_n(score_table, N, 1.0f / N );
    int outgoing_table[ N ] = { 0 };

    read_inputfile( visited_matrix, outgoing_table );
    
    float *dev_score_table, *dev_old_score_table;
    int *dev_outgoing_table;
    vis_m_col *dev_visited_matrix;
    trans_m_col *dev_transition_matrix;

    hipMalloc( &dev_score_table, score_t_size );
    hipMalloc( &dev_old_score_table, score_t_size );
    hipMalloc( &dev_outgoing_table, out_t_size );
    hipMalloc( &dev_visited_matrix, vis_m_size );
    hipMalloc( &dev_transition_matrix, trans_m_size );

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    
    hipMemcpy( dev_score_table, score_table, score_t_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_outgoing_table, outgoing_table, out_t_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_visited_matrix, visited_matrix, vis_m_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_transition_matrix, transition_matrix, trans_m_size, hipMemcpyHostToDevice );

    
    /* timing the PageRank algorithm */
    auto const pr_start_time = std::chrono::steady_clock::now();

    auto num_blocks = ceil( N * N / static_cast< float >( blocksize ) );
    update_entries<<< num_blocks, blocksize >>>( dev_transition_matrix, dev_visited_matrix, dev_outgoing_table, N );

    num_blocks = ceil( N / static_cast< float >( blocksize ) );
    /* iterations must be serial */
    for ( auto i = 0; i < num_iter - 1; ++i )
    {
        /* scores from previous iteration */
        hipMemcpy( dev_old_score_table, dev_score_table, score_t_size, hipMemcpyDeviceToDevice );
        pagerank<<< num_blocks, blocksize >>>( dev_score_table, dev_old_score_table, dev_transition_matrix, d, N );
    }

    /* retrieve final scores array from device and store back to host */
    hipMemcpy(score_table, dev_score_table, score_t_size, hipMemcpyDeviceToHost);

    auto const pr_end_time = std::chrono::steady_clock::now();
    auto const pr_time = std::chrono::duration_cast< std::chrono::microseconds >\
    ( pr_end_time - pr_start_time ).count();

    hipFree( dev_score_table );
    hipFree( dev_old_score_table );
    hipFree( dev_outgoing_table );
    hipFree( dev_visited_matrix );
    hipFree( dev_transition_matrix );

    auto const total_end_time = std::chrono::steady_clock::now();
    auto const total_time = std::chrono::duration_cast< std::chrono::microseconds >\
    ( total_end_time - total_start_time ).count();

    print_top_5( score_table );
    print_total( score_table );

    std::cout << "in_kernel_pagerank_time = "
              << pr_time
              << " us" 
              << "\nprogram_total_time = "
              << total_time
              << " us"
              << std::endl;
    
    return 0;
}
